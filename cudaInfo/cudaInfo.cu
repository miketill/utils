#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    printf("START\n");
    int device_count;
    hipGetDeviceCount(&device_count);
    printf("Number of CUDA devices: %d\n", device_count);

    hipDeviceProp_t props;
    for (int i = 0; i < device_count; i++) {
        hipGetDeviceProperties(&props, i);
        printf("Info for device #%d:\n ", i);
        printf("\tName: %s\n", props.name);
        printf("\tGlobal Memory: %d\n", props.totalGlobalMem);
        printf("\tShared Memory per Block: %d\n", props.sharedMemPerBlock);
        printf("\tRegisters per Block: %d\n", props.regsPerBlock);
        printf("\tWarp Size: %d\n", props.warpSize);
        printf("\tMax Threads per Block: %d\n", props.maxThreadsPerBlock);
        printf("\tClock Rate: %d\n", props.clockRate);
        printf("\tMemory Clock Rate: %d\n", props.memoryClockRate);
        printf("\tMemory Bus Width: %d\n", props.memoryBusWidth);
        printf("\tSM Count: %d\n", props.multiProcessorCount);
        printf("\tThreads per SM: %d\n", props.maxThreadsPerMultiProcessor);
        printf("\tConcurrent Kernels: %s\n", props.concurrentKernels ? "Y":"N");
        //printf("\t: %d\n", props.);
    }
    return 0;
}
